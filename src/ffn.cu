#include "hip/hip_runtime.h"
#include <random>
#include <cmath>
#include <algorithm>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime_api.h>

#include "ffn.h"

// Block width for CUDA kernel
// FIXME This is super naive
#define BW 128

__global__ void FillOnes(float *vec, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size)
        return;

    vec[idx] = 1.0f;
}

static inline unsigned int RoundUp(unsigned int nominator, unsigned int denominator)
{
    return (nominator + denominator - 1) / denominator;
}

__global__ void SoftmaxLossBackprop(const float *label, int num_labels, int batch_size, float *diff)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= batch_size)
        return;

    const int label_value = static_cast<int>(label[idx]);

    // For each item in the batch, decrease the result of the label's value by 1
    diff[idx * num_labels + label_value] -= 1.0f;
}

FullyConnectedLayer::FullyConnectedLayer(int inputs, int outputs)
    : inputs(inputs),
    outputs(outputs),
    neurons(inputs * outputs),
    bias(outputs) {}


TrainingContext::TrainingContext(int batch_size, FullyConnectedLayer& fc1, FullyConnectedLayer& fc2, std::default_random_engine rd, int train_size, int test_size)
    : batch_size(batch_size),
    fc1(fc1),
    fc2(fc2),
    rd(rd),
    train_size(train_size),
    test_size(test_size) {
        checkCudaErrors(hipSetDevice(0));

        checkCudaErrors(hipblasCreate(&cublas_handle));
        checkCUDNN(hipdnnCreate(&cudnn_handle));

        checkCUDNN(hipdnnCreateTensorDescriptor(&data_tensor));
        checkCUDNN(hipdnnCreateTensorDescriptor(&fc1_tensor));
        checkCUDNN(hipdnnCreateTensorDescriptor(&fc2_tensor));

        checkCUDNN(hipdnnCreateActivationDescriptor(&fc1_activation));

        checkCUDNN(hipdnnSetTensor4dDescriptor(fc1_tensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batch_size, fc1.outputs, 1, 1));
        checkCUDNN(hipdnnSetTensor4dDescriptor(fc2_tensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batch_size, fc2.outputs, 1, 1));
        checkCUDNN(hipdnnSetActivationDescriptor(fc1_activation, HIPDNN_ACTIVATION_SIGMOID, HIPDNN_NOT_PROPAGATE_NAN, 0.0));
    }

TrainingContext::~TrainingContext() {
    checkCudaErrors(hipblasDestroy(cublas_handle));
    checkCUDNN(hipdnnDestroy(cudnn_handle));

    checkCUDNN(hipdnnDestroyTensorDescriptor(data_tensor));
    checkCUDNN(hipdnnDestroyTensorDescriptor(fc1_tensor));
    checkCUDNN(hipdnnDestroyTensorDescriptor(fc2_tensor));
    checkCUDNN(hipdnnDestroyActivationDescriptor(fc1_activation));
}

void TrainingContext::weight_initialization() {
    // Xavier init
    float wfc1 = 4.0f * sqrt(6.0f / (fc1.inputs + fc1.outputs));
    std::uniform_real_distribution<> dfc1(-wfc1, wfc1);
    float wfc2 = 4.0f * sqrt(6.0f / (fc2.inputs + fc2.outputs));
    std::uniform_real_distribution<> dfc2(-wfc2, wfc2);

    std::generate_n(fc1.neurons.begin(), fc1.neurons.size(), [&]{ return dfc1(rd);});
    std::generate_n(fc2.neurons.begin(), fc2.neurons.size(), [&]{ return dfc2(rd);});

    std::generate_n(fc1.bias.begin(), fc1.bias.size(), [&]{ return dfc1(rd);});
    std::generate_n(fc2.bias.begin(), fc2.bias.size(), [&]{ return dfc2(rd);});
}

void TrainingContext::initialize(int channels, int height, int width) {
    this->channels = channels;
    this->height = height;
    this->width = width;
    // Memory allocation for the propagated data
    checkCudaErrors(hipMalloc(&d_data, sizeof(float) * this->batch_size * channels * height * width));
    checkCudaErrors(hipMalloc(&d_labels, sizeof(float) * this->batch_size));
    checkCudaErrors(hipMalloc(&d_fc1_pre, sizeof(float) * this->batch_size * fc1.outputs));
    checkCudaErrors(hipMalloc(&d_fc1_post, sizeof(float) * this-> batch_size * fc1.outputs));
    checkCudaErrors(hipMalloc(&d_fc2_pre, sizeof(float) * this->batch_size * fc2.outputs));
    checkCudaErrors(hipMalloc(&d_fc2_post, sizeof(float) * this->batch_size * fc2.outputs));

    // Memory allocation for the network parameters
    checkCudaErrors(hipMalloc(&w_fc1, sizeof(float) * fc1.neurons.size()));
    checkCudaErrors(hipMalloc(&w_fc1bias, sizeof(float) * fc1.bias.size()));
    checkCudaErrors(hipMalloc(&w_fc2, sizeof(float) * fc2.neurons.size()));
    checkCudaErrors(hipMalloc(&w_fc2bias, sizeof(float) * fc2.bias.size()));

    // Memory allocation for temporary data
    checkCudaErrors(hipMalloc(&d_onevec, sizeof(float) * batch_size));

    // Memory allocation for differentials
    checkCudaErrors(hipMalloc(&dd_fc1, sizeof(float) * batch_size * fc1.inputs));
    checkCudaErrors(hipMalloc(&dd_fc1_post, sizeof(float) * batch_size * fc1.outputs));
    checkCudaErrors(hipMalloc(&dd_fc2, sizeof(float) * batch_size * fc2.inputs));
    checkCudaErrors(hipMalloc(&dd_fc2_post, sizeof(float) * batch_size * fc2.outputs));
    checkCudaErrors(hipMalloc(&d_loss, sizeof(float) * batch_size * fc2.outputs));

    // Memory allocation for gradients of network parameters
    checkCudaErrors(hipMalloc(&dw_fc1, sizeof(float) * fc1.neurons.size()));
    checkCudaErrors(hipMalloc(&dw_fc1bias, sizeof(float) * fc1.bias.size()));
    checkCudaErrors(hipMalloc(&dw_fc2, sizeof(float) * fc2.neurons.size()));
    checkCudaErrors(hipMalloc(&dw_fc2bias, sizeof(float) * fc2.bias.size()));

    // Populate GPU global memory
    checkCudaErrors(hipMemcpyAsync(w_fc1, &fc1.neurons[0], sizeof(float) * fc1.neurons.size(), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpyAsync(w_fc1bias, &fc1.bias[0], sizeof(float) * fc1.bias.size(), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpyAsync(w_fc2, &fc2.neurons[0], sizeof(float) * fc2.neurons.size(), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpyAsync(w_fc2bias, &fc2.bias[0], sizeof(float) * fc2.bias.size(), hipMemcpyHostToDevice));

    FillOnes<<<RoundUp(batch_size, BW), BW>>>(d_onevec, batch_size);
}

void TrainingContext::forward() {
    float alpha = 1.0f, beta = 0.0f;

    // Input -> layer 1 propagation
    checkCudaErrors(hipblasSgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, fc1.outputs, batch_size, fc1.inputs, &alpha, w_fc1, fc1.inputs, d_data, fc1.inputs, &beta, d_fc1_pre, fc1.outputs));
    // Add layer 1 bias
    checkCudaErrors(hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, fc1.outputs, batch_size, 1, &alpha, w_fc1bias, fc1.outputs, d_onevec, 1, &alpha, d_fc1_pre, fc1.outputs));
    // Apply sigmoid activation
    checkCUDNN(hipdnnActivationForward(cudnn_handle, fc1_activation, &alpha, fc1_tensor, d_fc1_pre, &beta, fc1_tensor, d_fc1_post));

    // Layer 1 -> layer 2 propagation
    checkCudaErrors(hipblasSgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, fc2.outputs, batch_size, fc2.inputs, &alpha, w_fc2, fc2.inputs, d_fc1_post, fc2.inputs, &beta, d_fc2_pre, fc2.outputs));
    // Add layer 2 bias
    checkCudaErrors(hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, fc2.outputs, batch_size, 1, &alpha, w_fc2bias, fc2.outputs, d_onevec, 1, &alpha, d_fc2_pre, fc2.outputs));
    // Softmax loss
    checkCUDNN(hipdnnSoftmaxForward(cudnn_handle, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL, &alpha, fc2_tensor, d_fc2_pre, &beta, fc2_tensor, d_fc2_post));
}

void TrainingContext::backward() {
    float alpha = 1.0f, beta = 0.0f;

    // FIXME I don't really understand what this is
    float scale_value = 1.0f / static_cast<float>(batch_size);

    checkCudaErrors(hipMemcpyAsync(d_loss, d_fc2_post, sizeof(float) * batch_size * fc2.outputs, hipMemcpyDeviceToDevice));

    SoftmaxLossBackprop<<<RoundUp(batch_size, BW), BW>>>(d_labels, fc2.outputs, batch_size, d_loss);

    checkCudaErrors(hipblasSscal(cublas_handle, fc2.outputs * batch_size, &scale_value, d_loss, 1));

    // Layer 2
    // Derivative w.r.t. weights dw_fc2 = (d_fc1_post * d_fc2_post)
    checkCudaErrors(hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, fc2.inputs, fc2.outputs, batch_size, &alpha, d_fc1_post, fc2.inputs, d_loss, fc2.outputs, &beta, dw_fc2, fc2.inputs));
    // Derivative w.r.t. bias dw_fc2bias = d_loss * 1_vec
    checkCudaErrors(hipblasSgemv(cublas_handle, HIPBLAS_OP_N, fc2.outputs, batch_size, &alpha, d_loss, fc2.outputs, d_onevec, 1, &beta, dw_fc2bias, 1));
    // Derivative w.r.t. data d_fc2_pre * d_fc2_post
    checkCudaErrors(hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, fc2.inputs, batch_size, fc2.outputs, &alpha, w_fc2, fc2.inputs, d_loss, fc2.outputs, &beta, dd_fc2, fc2.inputs));
    checkCUDNN(hipdnnActivationBackward(cudnn_handle, fc1_activation, &alpha, fc1_tensor, d_fc1_post, fc1_tensor, dd_fc2, fc1_tensor, d_fc1_pre, &beta, fc1_tensor, dd_fc1_post));

    // Layer 1
    // Derivative w.r.t. weights d_g_fc1 = d_data * d_fc1_post)
    checkCudaErrors(hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, fc1.inputs, fc1.outputs, batch_size, &alpha, d_data, fc1.inputs, dd_fc1_post, fc1.outputs, &beta, dw_fc1, fc1.inputs));
    // Derivative w.r.t. bias dw_fc1bias = d_fc1_post * 1_vec
    checkCudaErrors(hipblasSgemv(cublas_handle, HIPBLAS_OP_N, fc1.outputs, batch_size, &alpha, dd_fc1_post, fc1.outputs, d_onevec, 1, &beta, dw_fc1bias, 1));
}

void TrainingContext::update(float learning_rate) {
    float alpha = -learning_rate;

    checkCudaErrors(hipblasSaxpy(cublas_handle, static_cast<int>(fc1.neurons.size()), &alpha, dw_fc1, 1, w_fc1, 1));
    checkCudaErrors(hipblasSaxpy(cublas_handle, static_cast<int>(fc1.bias.size()), &alpha, dw_fc1bias, 1, w_fc1bias, 1));

    checkCudaErrors(hipblasSaxpy(cublas_handle, static_cast<int>(fc2.neurons.size()), &alpha, dw_fc2, 1, w_fc2, 1));
    checkCudaErrors(hipblasSaxpy(cublas_handle, static_cast<int>(fc2.bias.size()), &alpha, dw_fc2bias, 1, w_fc2bias, 1));
}

void TrainingContext::train(int iter) {
    checkCudaErrors(hipDeviceSynchronize());
    for (int i = 0; i < iter; ++i) {
        int imageid = i % (train_size / batch_size);

        // Copy current batch to GPU
        checkCudaErrors(hipMemcpyAsync(d_data, &train_images[imageid * batch_size * width * height * channels], sizeof(float) * batch_size * channels * width * height, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpyAsync(d_labels, &train_labels[imageid * batch_size], sizeof(float) * batch_size, hipMemcpyHostToDevice));

        forward();

        backward();

        float learning_rate = static_cast<float>(0.01 * pow((1.0 + 0.0001 * i), (-0.75)));
        update(learning_rate);

        if (i % 1000 == 0) {
            std::cout << ".";
            fflush(stdout);
        }
    }
    checkCudaErrors(hipDeviceSynchronize());
}

void TrainingContext::test() {
    checkCudaErrors(hipDeviceSynchronize());

    int num_errors = 0;

    for (int i = 0; i < test_size; ++i) {
        // We'll test images one by one (instead of mini-batch style)
        std::vector<float> data(width * height);
        for (int j = 0; j < width * height; ++j) {
            data[j] = (float)test_images[i * width * height * channels + j];
        }

        checkCudaErrors(hipMemcpyAsync(d_data, &data[0], sizeof(float) * channels * width * height, hipMemcpyHostToDevice));

        forward();

        std::vector<float> class_vector(10);
        checkCudaErrors(hipMemcpy(&class_vector[0], d_fc2_post, sizeof(float) * 10, hipMemcpyDeviceToHost));

        int chosen = 0;
        for (int id = 1; id < 10; ++id) {
            if (class_vector[chosen] < class_vector[id]) {
                chosen = id;
            }
        }

        if (chosen != test_labels[i]) {
            num_errors++;
        }
    }
    float total_error = (float) num_errors / (float) test_size;
    printf("Classification result: %.2f%% error (used %d images)\n", total_error * 100.0f, (int) test_size);
}

void TrainingContext::destroy() {
    checkCudaErrors(hipFree(d_data));
    checkCudaErrors(hipFree(d_labels));
    checkCudaErrors(hipFree(d_fc1_pre));
    checkCudaErrors(hipFree(d_fc1_post));
    checkCudaErrors(hipFree(d_fc2_pre));
    checkCudaErrors(hipFree(d_fc2_post));

    checkCudaErrors(hipFree(w_fc1));
    checkCudaErrors(hipFree(w_fc1bias));
    checkCudaErrors(hipFree(w_fc2));
    checkCudaErrors(hipFree(w_fc2bias));

    checkCudaErrors(hipFree(d_onevec));

    checkCudaErrors(hipFree(dd_fc1));
    checkCudaErrors(hipFree(dd_fc1_post));
    checkCudaErrors(hipFree(dd_fc2));
    checkCudaErrors(hipFree(dd_fc2_post));
    checkCudaErrors(hipFree(d_loss));

    checkCudaErrors(hipFree(dw_fc1));
    checkCudaErrors(hipFree(dw_fc1bias));
    checkCudaErrors(hipFree(dw_fc2));
    checkCudaErrors(hipFree(dw_fc2bias));
}
