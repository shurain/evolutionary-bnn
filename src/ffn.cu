#include "hip/hip_runtime.h"
#include <random>
#include <cmath>
#include <algorithm>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime_api.h>

#include "ffn.h"

// Block width for CUDA kernel
// FIXME This is super naive
#define BW 128

__global__ void FillOnes(float *vec, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size)
        return;

    vec[idx] = 1.0f;
}

static inline unsigned int RoundUp(unsigned int nominator, unsigned int denominator)
{
    return (nominator + denominator - 1) / denominator;
}

__global__ void SoftmaxLossBackprop(const float *label, int num_labels, int batch_size, float *diff)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= batch_size)
        return;

    const int label_value = static_cast<int>(label[idx]);

    // For each item in the batch, decrease the result of the label's value by 1
    diff[idx * num_labels + label_value] -= 1.0f;
}

FullyConnectedLayer::FullyConnectedLayer(int inputs, int outputs)
    : inputs(inputs),
    outputs(outputs),
    neurons(inputs * outputs),
    bias(outputs) {}


TrainingContext::TrainingContext(int batch_size, FullyConnectedLayer& fc1, FullyConnectedLayer& fc2, FullyConnectedLayer& fc3, FullyConnectedLayer& fc4, std::default_random_engine rd, int train_size, int test_size)
    : batch_size(batch_size),
    fc1(fc1),
    fc2(fc2),
    fc3(fc3),
    fc4(fc4),
    rd(rd),
    train_size(train_size),
    test_size(test_size) {
        checkCudaErrors(hipSetDevice(0));

        checkCudaErrors(hipblasCreate(&cublas_handle));
        checkCUDNN(hipdnnCreate(&cudnn_handle));

        checkCUDNN(hipdnnCreateTensorDescriptor(&data_tensor));
        checkCUDNN(hipdnnCreateTensorDescriptor(&fc1_tensor));
        checkCUDNN(hipdnnCreateTensorDescriptor(&fc2_tensor));
        checkCUDNN(hipdnnCreateTensorDescriptor(&fc3_tensor));
        checkCUDNN(hipdnnCreateTensorDescriptor(&fc4_tensor));

        checkCUDNN(hipdnnCreateActivationDescriptor(&fc1_activation));
        checkCUDNN(hipdnnCreateActivationDescriptor(&fc2_activation));
        checkCUDNN(hipdnnCreateActivationDescriptor(&fc3_activation));

        checkCUDNN(hipdnnSetTensor4dDescriptor(fc1_tensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batch_size, fc1.outputs, 1, 1));
        checkCUDNN(hipdnnSetTensor4dDescriptor(fc2_tensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batch_size, fc2.outputs, 1, 1));
        checkCUDNN(hipdnnSetTensor4dDescriptor(fc3_tensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batch_size, fc3.outputs, 1, 1));
        checkCUDNN(hipdnnSetTensor4dDescriptor(fc4_tensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batch_size, fc4.outputs, 1, 1));
        checkCUDNN(hipdnnSetActivationDescriptor(fc1_activation, HIPDNN_ACTIVATION_SIGMOID, HIPDNN_NOT_PROPAGATE_NAN, 0.0));
        checkCUDNN(hipdnnSetActivationDescriptor(fc2_activation, HIPDNN_ACTIVATION_SIGMOID, HIPDNN_NOT_PROPAGATE_NAN, 0.0));
        checkCUDNN(hipdnnSetActivationDescriptor(fc3_activation, HIPDNN_ACTIVATION_SIGMOID, HIPDNN_NOT_PROPAGATE_NAN, 0.0));
    }

TrainingContext::~TrainingContext() {
    checkCudaErrors(hipblasDestroy(cublas_handle));
    checkCUDNN(hipdnnDestroy(cudnn_handle));

    checkCUDNN(hipdnnDestroyTensorDescriptor(data_tensor));
    checkCUDNN(hipdnnDestroyTensorDescriptor(fc1_tensor));
    checkCUDNN(hipdnnDestroyTensorDescriptor(fc2_tensor));
    checkCUDNN(hipdnnDestroyTensorDescriptor(fc3_tensor));
    checkCUDNN(hipdnnDestroyTensorDescriptor(fc4_tensor));
    checkCUDNN(hipdnnDestroyActivationDescriptor(fc1_activation));
    checkCUDNN(hipdnnDestroyActivationDescriptor(fc2_activation));
    checkCUDNN(hipdnnDestroyActivationDescriptor(fc3_activation));
}

void TrainingContext::weight_initialization() {
    // Xavier init
    float wfc1 = 4.0f * sqrt(6.0f / (fc1.inputs + fc1.outputs));
    std::uniform_real_distribution<> dfc1(-wfc1, wfc1);
    float wfc2 = 4.0f * sqrt(6.0f / (fc2.inputs + fc2.outputs));
    std::uniform_real_distribution<> dfc2(-wfc2, wfc2);
    float wfc3 = 4.0f * sqrt(6.0f / (fc3.inputs + fc3.outputs));
    std::uniform_real_distribution<> dfc3(-wfc3, wfc3);
    float wfc4 = 4.0f * sqrt(6.0f / (fc4.inputs + fc4.outputs));
    std::uniform_real_distribution<> dfc4(-wfc4, wfc4);

    std::generate_n(fc1.neurons.begin(), fc1.neurons.size(), [&]{ return dfc1(rd);});
    std::generate_n(fc2.neurons.begin(), fc2.neurons.size(), [&]{ return dfc2(rd);});
    std::generate_n(fc3.neurons.begin(), fc3.neurons.size(), [&]{ return dfc3(rd);});
    std::generate_n(fc4.neurons.begin(), fc4.neurons.size(), [&]{ return dfc4(rd);});

    // FIXME Usually we initialize bias as zeros
    std::generate_n(fc1.bias.begin(), fc1.bias.size(), [&]{ return dfc1(rd);});
    std::generate_n(fc2.bias.begin(), fc2.bias.size(), [&]{ return dfc2(rd);});
    std::generate_n(fc3.bias.begin(), fc3.bias.size(), [&]{ return dfc3(rd);});
    std::generate_n(fc4.bias.begin(), fc4.bias.size(), [&]{ return dfc4(rd);});
}

void TrainingContext::initialize(int channels, int height, int width) {
    this->channels = channels;
    this->height = height;
    this->width = width;
    // Memory allocation for the propagated data
    checkCudaErrors(hipMalloc(&d_data, sizeof(float) * this->batch_size * channels * height * width));
    checkCudaErrors(hipMalloc(&d_labels, sizeof(float) * this->batch_size));
    checkCudaErrors(hipMalloc(&d_fc1_pre, sizeof(float) * this->batch_size * fc1.outputs));
    checkCudaErrors(hipMalloc(&d_fc1_post, sizeof(float) * this-> batch_size * fc1.outputs));
    checkCudaErrors(hipMalloc(&d_fc2_pre, sizeof(float) * this->batch_size * fc2.outputs));
    checkCudaErrors(hipMalloc(&d_fc2_post, sizeof(float) * this->batch_size * fc2.outputs));
    checkCudaErrors(hipMalloc(&d_fc3_pre, sizeof(float) * this->batch_size * fc3.outputs));
    checkCudaErrors(hipMalloc(&d_fc3_post, sizeof(float) * this->batch_size * fc3.outputs));
    checkCudaErrors(hipMalloc(&d_fc4_pre, sizeof(float) * this->batch_size * fc4.outputs));
    checkCudaErrors(hipMalloc(&d_fc4_post, sizeof(float) * this->batch_size * fc4.outputs));

    // Memory allocation for the network parameters
    checkCudaErrors(hipMalloc(&w_fc1, sizeof(float) * fc1.neurons.size()));
    checkCudaErrors(hipMalloc(&w_fc1bias, sizeof(float) * fc1.bias.size()));
    checkCudaErrors(hipMalloc(&w_fc2, sizeof(float) * fc2.neurons.size()));
    checkCudaErrors(hipMalloc(&w_fc2bias, sizeof(float) * fc2.bias.size()));
    checkCudaErrors(hipMalloc(&w_fc3, sizeof(float) * fc3.neurons.size()));
    checkCudaErrors(hipMalloc(&w_fc3bias, sizeof(float) * fc3.bias.size()));
    checkCudaErrors(hipMalloc(&w_fc4, sizeof(float) * fc4.neurons.size()));
    checkCudaErrors(hipMalloc(&w_fc4bias, sizeof(float) * fc4.bias.size()));

    // Memory allocation for temporary data
    checkCudaErrors(hipMalloc(&d_onevec, sizeof(float) * batch_size));

    // Memory allocation for differentials
    checkCudaErrors(hipMalloc(&dd_fc1, sizeof(float) * batch_size * fc1.inputs));
    checkCudaErrors(hipMalloc(&dd_fc1_post, sizeof(float) * batch_size * fc1.outputs));
    checkCudaErrors(hipMalloc(&dd_fc2, sizeof(float) * batch_size * fc2.inputs));
    checkCudaErrors(hipMalloc(&dd_fc2_post, sizeof(float) * batch_size * fc2.outputs));
    checkCudaErrors(hipMalloc(&dd_fc3, sizeof(float) * batch_size * fc3.inputs));
    checkCudaErrors(hipMalloc(&dd_fc3_post, sizeof(float) * batch_size * fc3.outputs));
    checkCudaErrors(hipMalloc(&dd_fc4, sizeof(float) * batch_size * fc4.inputs));
    checkCudaErrors(hipMalloc(&dd_fc4_post, sizeof(float) * batch_size * fc4.outputs));
    checkCudaErrors(hipMalloc(&d_loss, sizeof(float) * batch_size * fc4.outputs));

    // Memory allocation for gradients of network parameters
    checkCudaErrors(hipMalloc(&dw_fc1, sizeof(float) * fc1.neurons.size()));
    checkCudaErrors(hipMalloc(&dw_fc1bias, sizeof(float) * fc1.bias.size()));
    checkCudaErrors(hipMalloc(&dw_fc2, sizeof(float) * fc2.neurons.size()));
    checkCudaErrors(hipMalloc(&dw_fc2bias, sizeof(float) * fc2.bias.size()));
    checkCudaErrors(hipMalloc(&dw_fc3, sizeof(float) * fc3.neurons.size()));
    checkCudaErrors(hipMalloc(&dw_fc3bias, sizeof(float) * fc3.bias.size()));
    checkCudaErrors(hipMalloc(&dw_fc4, sizeof(float) * fc4.neurons.size()));
    checkCudaErrors(hipMalloc(&dw_fc4bias, sizeof(float) * fc4.bias.size()));

    // Populate GPU global memory
    checkCudaErrors(hipMemcpyAsync(w_fc1, &fc1.neurons[0], sizeof(float) * fc1.neurons.size(), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpyAsync(w_fc1bias, &fc1.bias[0], sizeof(float) * fc1.bias.size(), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpyAsync(w_fc2, &fc2.neurons[0], sizeof(float) * fc2.neurons.size(), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpyAsync(w_fc2bias, &fc2.bias[0], sizeof(float) * fc2.bias.size(), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpyAsync(w_fc3, &fc3.neurons[0], sizeof(float) * fc3.neurons.size(), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpyAsync(w_fc3bias, &fc3.bias[0], sizeof(float) * fc3.bias.size(), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpyAsync(w_fc4, &fc4.neurons[0], sizeof(float) * fc4.neurons.size(), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpyAsync(w_fc4bias, &fc4.bias[0], sizeof(float) * fc4.bias.size(), hipMemcpyHostToDevice));

    FillOnes<<<RoundUp(batch_size, BW), BW>>>(d_onevec, batch_size);
}

void TrainingContext::forward() {
    float alpha = 1.0f, beta = 0.0f;

    // Input -> layer 1 propagation
    checkCudaErrors(hipblasSgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, fc1.outputs, batch_size, fc1.inputs, &alpha, w_fc1, fc1.inputs, d_data, fc1.inputs, &beta, d_fc1_pre, fc1.outputs));
    // Add layer 1 bias
    checkCudaErrors(hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, fc1.outputs, batch_size, 1, &alpha, w_fc1bias, fc1.outputs, d_onevec, 1, &alpha, d_fc1_pre, fc1.outputs));
    // Apply sigmoid activation
    checkCUDNN(hipdnnActivationForward(cudnn_handle, fc1_activation, &alpha, fc1_tensor, d_fc1_pre, &beta, fc1_tensor, d_fc1_post));

    // Layer 1 -> layer 2 propagation
    checkCudaErrors(hipblasSgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, fc2.outputs, batch_size, fc2.inputs, &alpha, w_fc2, fc2.inputs, d_fc1_post, fc2.inputs, &beta, d_fc2_pre, fc2.outputs));
    // Add layer 2 bias
    checkCudaErrors(hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, fc2.outputs, batch_size, 1, &alpha, w_fc2bias, fc2.outputs, d_onevec, 1, &alpha, d_fc2_pre, fc2.outputs));
    // Apply sigmoid activation
    checkCUDNN(hipdnnActivationForward(cudnn_handle, fc2_activation, &alpha, fc2_tensor, d_fc2_pre, &beta, fc2_tensor, d_fc2_post));

    // Layer 2 -> layer 3 propagation
    checkCudaErrors(hipblasSgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, fc3.outputs, batch_size, fc3.inputs, &alpha, w_fc3, fc3.inputs, d_fc2_post, fc3.inputs, &beta, d_fc3_pre, fc3.outputs));
    // Add layer 3 bias
    checkCudaErrors(hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, fc3.outputs, batch_size, 1, &alpha, w_fc3bias, fc3.outputs, d_onevec, 1, &alpha, d_fc3_pre, fc3.outputs));
    // Apply sigmoid activation
    checkCUDNN(hipdnnActivationForward(cudnn_handle, fc3_activation, &alpha, fc3_tensor, d_fc3_pre, &beta, fc3_tensor, d_fc3_post));

    // Layer 3 -> Output layer propagation
    checkCudaErrors(hipblasSgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, fc4.outputs, batch_size, fc4.inputs, &alpha, w_fc4, fc4.inputs, d_fc3_post, fc4.inputs, &beta, d_fc4_pre, fc4.outputs));
    // Add Output layer bias
    checkCudaErrors(hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, fc4.outputs, batch_size, 1, &alpha, w_fc4bias, fc4.outputs, d_onevec, 1, &alpha, d_fc4_pre, fc4.outputs));
    // Softmax loss
    checkCUDNN(hipdnnSoftmaxForward(cudnn_handle, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL, &alpha, fc4_tensor, d_fc4_pre, &beta, fc4_tensor, d_fc4_post));
}

void TrainingContext::backward() {
    float alpha = 1.0f, beta = 0.0f;

    // FIXME I don't really understand what this is
    float scale_value = 1.0f / static_cast<float>(batch_size);

    // Output layer
    checkCudaErrors(hipMemcpyAsync(d_loss, d_fc4_post, sizeof(float) * batch_size * fc4.outputs, hipMemcpyDeviceToDevice));
    SoftmaxLossBackprop<<<RoundUp(batch_size, BW), BW>>>(d_labels, fc4.outputs, batch_size, d_loss);
    checkCudaErrors(hipblasSscal(cublas_handle, fc4.outputs * batch_size, &scale_value, d_loss, 1));

    // Output -> Layer 3
    // Derivative w.r.t. weights dw_fc4 = (d_fc3_post * d_loss)
    checkCudaErrors(hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, fc4.inputs, fc4.outputs, batch_size, &alpha, d_fc3_post, fc4.inputs, d_loss, fc4.outputs, &beta, dw_fc4, fc4.inputs));
    // Derivative w.r.t. bias dw_fc4bias = d_loss * 1_vec
    checkCudaErrors(hipblasSgemv(cublas_handle, HIPBLAS_OP_N, fc4.outputs, batch_size, &alpha, d_loss, fc4.outputs, d_onevec, 1, &beta, dw_fc4bias, 1));
    // Derivative w.r.t. data dd_fc4 = w_fc4 * d_loss
    checkCudaErrors(hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, fc4.inputs, batch_size, fc4.outputs, &alpha, w_fc4, fc4.inputs, d_loss, fc4.outputs, &beta, dd_fc4, fc4.inputs));
    checkCUDNN(hipdnnActivationBackward(cudnn_handle, fc3_activation, &alpha, fc3_tensor, d_fc3_post, fc3_tensor, dd_fc4, fc3_tensor, d_fc3_pre, &beta, fc3_tensor, dd_fc3_post));

    // Layer 3 -> Layer 2
    // Derivative w.r.t. weights
    checkCudaErrors(hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, fc3.inputs, fc3.outputs, batch_size, &alpha, d_fc2_post, fc3.inputs, dd_fc3_post, fc3.outputs, &beta, dw_fc3, fc3.inputs));
    // Derivative w.r.t. bias
    checkCudaErrors(hipblasSgemv(cublas_handle, HIPBLAS_OP_N, fc3.outputs, batch_size, &alpha, dd_fc3_post, fc3.outputs, d_onevec, 1, &beta, dw_fc3bias, 1));
    // Derivative w.r.t. data
    checkCudaErrors(hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, fc3.inputs, batch_size, fc3.outputs, &alpha, w_fc3, fc3.inputs, dd_fc3_post, fc3.outputs, &beta, dd_fc3, fc3.inputs));
    checkCUDNN(hipdnnActivationBackward(cudnn_handle, fc2_activation, &alpha, fc2_tensor, d_fc2_post, fc2_tensor, dd_fc3, fc2_tensor, d_fc2_pre, &beta, fc2_tensor, dd_fc2_post));

    // Layer 2 -> Layer 1
    // Derivative w.r.t. weights
    checkCudaErrors(hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, fc2.inputs, fc2.outputs, batch_size, &alpha, d_fc1_post, fc2.inputs, dd_fc2_post, fc2.outputs, &beta, dw_fc2, fc2.inputs));
    // Derivative w.r.t. bias
    checkCudaErrors(hipblasSgemv(cublas_handle, HIPBLAS_OP_N, fc2.outputs, batch_size, &alpha, dd_fc2_post, fc2.outputs, d_onevec, 1, &beta, dw_fc2bias, 1));
    // Derivative w.r.t. data
    checkCudaErrors(hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, fc2.inputs, batch_size, fc2.outputs, &alpha, w_fc2, fc2.inputs, dd_fc2_post, fc2.outputs, &beta, dd_fc2, fc2.inputs));
    checkCUDNN(hipdnnActivationBackward(cudnn_handle, fc1_activation, &alpha, fc1_tensor, d_fc1_post, fc1_tensor, dd_fc2, fc1_tensor, d_fc1_pre, &beta, fc1_tensor, dd_fc1_post));

    // Layer 1
    // Derivative w.r.t. weights
    checkCudaErrors(hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, fc1.inputs, fc1.outputs, batch_size, &alpha, d_data, fc1.inputs, dd_fc1_post, fc1.outputs, &beta, dw_fc1, fc1.inputs));
    // Derivative w.r.t. bias
    checkCudaErrors(hipblasSgemv(cublas_handle, HIPBLAS_OP_N, fc1.outputs, batch_size, &alpha, dd_fc1_post, fc1.outputs, d_onevec, 1, &beta, dw_fc1bias, 1));
    // No need to compute derivatives w.r.t. data
}

void TrainingContext::update(float learning_rate) {
    float alpha = -learning_rate;

    checkCudaErrors(hipblasSaxpy(cublas_handle, static_cast<int>(fc1.neurons.size()), &alpha, dw_fc1, 1, w_fc1, 1));
    checkCudaErrors(hipblasSaxpy(cublas_handle, static_cast<int>(fc1.bias.size()), &alpha, dw_fc1bias, 1, w_fc1bias, 1));

    checkCudaErrors(hipblasSaxpy(cublas_handle, static_cast<int>(fc2.neurons.size()), &alpha, dw_fc2, 1, w_fc2, 1));
    checkCudaErrors(hipblasSaxpy(cublas_handle, static_cast<int>(fc2.bias.size()), &alpha, dw_fc2bias, 1, w_fc2bias, 1));

    checkCudaErrors(hipblasSaxpy(cublas_handle, static_cast<int>(fc3.neurons.size()), &alpha, dw_fc3, 1, w_fc3, 1));
    checkCudaErrors(hipblasSaxpy(cublas_handle, static_cast<int>(fc3.bias.size()), &alpha, dw_fc3bias, 1, w_fc3bias, 1));

    checkCudaErrors(hipblasSaxpy(cublas_handle, static_cast<int>(fc4.neurons.size()), &alpha, dw_fc4, 1, w_fc4, 1));
    checkCudaErrors(hipblasSaxpy(cublas_handle, static_cast<int>(fc4.bias.size()), &alpha, dw_fc4bias, 1, w_fc4bias, 1));
}

void TrainingContext::train(int iter) {
    checkCudaErrors(hipDeviceSynchronize());
    for (int i = 0; i < iter; ++i) {
        int imageid = i % (train_size / batch_size);

        // Copy current batch to GPU
        checkCudaErrors(hipMemcpyAsync(d_data, &train_images[imageid * batch_size * width * height * channels], sizeof(float) * batch_size * channels * width * height, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpyAsync(d_labels, &train_labels[imageid * batch_size], sizeof(float) * batch_size, hipMemcpyHostToDevice));

        forward();

        backward();

        float learning_rate = static_cast<float>(0.01 * pow((1.0 + 0.0001 * i), (-0.75)));
        update(learning_rate);

        if (i % 1000 == 0) {
            std::cout << ".";
            fflush(stdout);
        }
    }
    checkCudaErrors(hipDeviceSynchronize());
}

void TrainingContext::test() {
    checkCudaErrors(hipDeviceSynchronize());

    int num_errors = 0;

    for (int i = 0; i < test_size; ++i) {
        // We'll test images one by one (instead of mini-batch style)
        std::vector<float> data(width * height);
        for (int j = 0; j < width * height; ++j) {
            data[j] = (float)test_images[i * width * height * channels + j];
        }

        checkCudaErrors(hipMemcpyAsync(d_data, &data[0], sizeof(float) * channels * width * height, hipMemcpyHostToDevice));

        forward();

        std::vector<float> class_vector(10);
        checkCudaErrors(hipMemcpy(&class_vector[0], d_fc4_post, sizeof(float) * 10, hipMemcpyDeviceToHost));

        int chosen = 0;
        for (int id = 1; id < 10; ++id) {
            if (class_vector[chosen] < class_vector[id]) {
                chosen = id;
            }
        }

        if (chosen != test_labels[i]) {
            num_errors++;
        }
    }
    float total_error = (float) num_errors / (float) test_size;
    printf("Classification result: %.2f%% error (used %d images)\n", total_error * 100.0f, (int) test_size);
}

void TrainingContext::destroy() {
    checkCudaErrors(hipFree(d_data));
    checkCudaErrors(hipFree(d_labels));
    checkCudaErrors(hipFree(d_fc1_pre));
    checkCudaErrors(hipFree(d_fc1_post));
    checkCudaErrors(hipFree(d_fc2_pre));
    checkCudaErrors(hipFree(d_fc2_post));
    checkCudaErrors(hipFree(d_fc3_pre));
    checkCudaErrors(hipFree(d_fc3_post));
    checkCudaErrors(hipFree(d_fc4_pre));
    checkCudaErrors(hipFree(d_fc4_post));

    checkCudaErrors(hipFree(w_fc1));
    checkCudaErrors(hipFree(w_fc1bias));
    checkCudaErrors(hipFree(w_fc2));
    checkCudaErrors(hipFree(w_fc2bias));
    checkCudaErrors(hipFree(w_fc3));
    checkCudaErrors(hipFree(w_fc3bias));
    checkCudaErrors(hipFree(w_fc4));
    checkCudaErrors(hipFree(w_fc4bias));

    checkCudaErrors(hipFree(d_onevec));

    checkCudaErrors(hipFree(dd_fc1));
    checkCudaErrors(hipFree(dd_fc1_post));
    checkCudaErrors(hipFree(dd_fc2));
    checkCudaErrors(hipFree(dd_fc2_post));
    checkCudaErrors(hipFree(dd_fc3));
    checkCudaErrors(hipFree(dd_fc3_post));
    checkCudaErrors(hipFree(dd_fc4));
    checkCudaErrors(hipFree(dd_fc4_post));
    checkCudaErrors(hipFree(d_loss));

    checkCudaErrors(hipFree(dw_fc1));
    checkCudaErrors(hipFree(dw_fc1bias));
    checkCudaErrors(hipFree(dw_fc2));
    checkCudaErrors(hipFree(dw_fc2bias));
    checkCudaErrors(hipFree(dw_fc3));
    checkCudaErrors(hipFree(dw_fc3bias));
    checkCudaErrors(hipFree(dw_fc4));
    checkCudaErrors(hipFree(dw_fc4bias));
}
